#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/compress_conv_dw_layer.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {

template <typename Dtype>
__global__ void CConvolutionDepthwiseWeightForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const weight_data,
    const int num, const int channels, const int top_height,
    const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / channels / top_height / top_width;
    const int c = (index / top_height / top_width) % channels;
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const Dtype* weight = weight_data + c * kernel_h * kernel_w;
    Dtype value = 0;
    for (int kh = 0; kh < kernel_h; ++kh) {
      for (int kw = 0; kw < kernel_w; ++kw) {
        const int h_in = -pad_h + h * stride_h + kh * dilation_h;
        const int w_in = -pad_w + w * stride_w + kw * dilation_w;
        if ((h_in >= 0) && (h_in < bottom_height)
              && (w_in >= 0) && (w_in < bottom_width)) {
          const int offset = ((n * channels + c) * bottom_height + h_in)
                * bottom_width + w_in;
          value += (*weight) * bottom_data[offset];
        }
        ++weight;
      }
    }
    top_data[index] = value;
  }
}

template <typename Dtype>
__global__ void CConvolutionDepthwiseBiasForward(const int nthreads,
    const Dtype* const bias_data, const int num, const int channels,
    const int top_height, const int top_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = (index / top_height / top_width) % channels;
    top_data[index] += bias_data[c];
  }
}

template <typename Dtype>
__global__ void CCMaskApply(const int n, const Dtype* wb,
    const Dtype* mask, Dtype* wb_t) {
  CUDA_KERNEL_LOOP(index, n) {
    wb_t[index] = wb[index] * mask[index];    
  }
}


template <typename Dtype>
void CConvolutionDepthwiseLayer<Dtype>::Forward_gpu(
      const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  //const Dtype* weight_data = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();
  const int num = top[0]->num();
  const int channels = top[0]->channels();
  const int top_height = top[0]->height();
  const int top_width = top[0]->width();
  const int bottom_height = bottom[0]->height();
  const int bottom_width = bottom[0]->width();
  
  size_t blob_num = this->blobs_.size();
  Dtype* weightTmp = this->weight_tmp_.mutable_gpu_data(); 
  
  if (this->phase_ == TRAIN){
		// Calculate the mean and standard deviation of learnable parameters 		
    if (this->iter_ % this->inter_iter_ == 0 && (this->iter_) < (this->iter_stop_) && this->is_pruning_){      
  		Dtype *weightMaskCPU = this->blobs_[blob_num - 1]->mutable_cpu_data();
  		Dtype *weightCPU = this->blobs_[0]->mutable_cpu_data();
        // compute the weight mask based on the inter_inter
        Dtype sparsity_ratio = this->bound_weight_ * log(2 + (this->iter_ / this->inter_iter_));
		// compute the mask
		caffe_set(this->blobs_[2]->count(), (Dtype)1.0, weightMaskCPU);
		vector<std::pair <Dtype, size_t> > param_temp;
		for (size_t i = 0; i < this->blobs_[2]->count(); i++)
			param_temp.push_back(std::make_pair(fabs(weightCPU[i]), i));

		std::sort(param_temp.begin(), param_temp.end(), sortPairAscend);
		for (size_t i = 0; i < this->blobs_[0]->count() * sparsity_ratio; i++)
			weightMaskCPU[param_temp[i].second] = 0.0;

		LOG(INFO) << sparsity_ratio << " " << param_temp[0].first<< " " << param_temp[this->blobs_[0]->count() - 1].first;

	}
		
  }   

  const Dtype* weight_data = this->blobs_[0]->mutable_gpu_data();  
  Dtype* weightMask = this->blobs_[blob_num - 1]->mutable_gpu_data();
 
  // Calculate the current (masked) weight and bias
  CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[0]->count()),
    CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[0]->count(), weight_data, weightMask, weightTmp);
  CUDA_POST_KERNEL_CHECK;
  CConvolutionDepthwiseWeightForward<Dtype>
        // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weightTmp, num, channels,
      top_height, top_width, bottom_height, bottom_width,
      kernel_h_, kernel_w_, stride_h_, stride_w_,
      pad_h_, pad_w_, dilation_h_, dilation_w_, top_data);
  if (this->layer_param_.convolution_param().bias_term()) {
    const Dtype* bias_data = this->blobs_[1]->gpu_data();
    CConvolutionDepthwiseBiasForward<Dtype>
          // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bias_data, num, channels,
        top_height, top_width, top_data);
  }
}

template <typename Dtype>
__global__ void CConvolutionDepthwiseWeightBackward(const int nthreads,
    const Dtype* const top_diff, const Dtype* const bottom_data,
    const int num, const int channels, const int top_height,
    const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, Dtype* const buffer_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const int kh = (index / kernel_w / num / top_height / top_width)
          % kernel_h;
    const int kw = (index / num / top_height / top_width) % kernel_w;
    const int h_in = -pad_h + h * stride_h + kh * dilation_h;
    const int w_in = -pad_w + w * stride_w + kw * dilation_w;
    if ((h_in >= 0) && (h_in < bottom_height)
          && (w_in >= 0) && (w_in < bottom_width)) {
      const int c = index / kernel_h / kernel_w / num / top_height / top_width;
      const int n = (index / top_height / top_width) % num;
      const int top_offset = ((n * channels + c) * top_height + h)
            * top_width + w;
      const int bottom_offset = ((n * channels + c) * bottom_height + h_in)
            * bottom_width + w_in;
      buffer_data[index] = top_diff[top_offset] * bottom_data[bottom_offset];
    } else {
      buffer_data[index] = 0;
    }
  }
}

template <typename Dtype>
__global__ void CConvolutionDepthwiseBottomBackward(const int nthreads,
    const Dtype* const top_diff, const Dtype* const weight_data,
    const int num, const int channels, const int top_height,
    const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / channels / bottom_height / bottom_width;
    const int c = (index / bottom_height / bottom_width) % channels;
    const int h = (index / bottom_width) % bottom_height;
    const int w = index % bottom_width;
    const Dtype* weight = weight_data + c * kernel_h * kernel_w;
    Dtype value = 0;
    for (int kh = 0; kh < kernel_h; ++kh) {
      for (int kw = 0; kw < kernel_w; ++kw) {
        const int h_out_s = h + pad_h - kh * dilation_h;
        const int w_out_s = w + pad_w - kw * dilation_w;
        if (((h_out_s % stride_h) == 0) && ((w_out_s % stride_w) == 0)) {
          const int h_out = h_out_s / stride_h;
          const int w_out = w_out_s / stride_w;
          if ((h_out >= 0) && (h_out < top_height)
                && (w_out >= 0) && (w_out < top_width)) {
            const int offset = ((n * channels + c) * top_height + h_out)
                  * top_width + w_out;
            value += (*weight) * top_diff[offset];
          }
        }
        ++weight;
      }
    }
    bottom_diff[index] += value;
  }
}

template <typename Dtype>
__global__ void CConvolutionDepthwiseBiasBackward(const int nthreads,
    const Dtype* const top_diff, const int num, const int channels,
    const int top_height, const int top_width, Dtype* const buffer_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = index / num / top_height / top_width;
    const int n = (index / top_height / top_width) % num;
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const int offset = ((n * channels + c) * top_height + h) * top_width + w;
    buffer_data[index] = top_diff[offset];
  }
}

template <typename Dtype>
void CConvolutionDepthwiseLayer<Dtype>::Backward_gpu(
      const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
      const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const int bottom_count = bottom[0]->count();
  const int num = top[0]->num();
  const int channels = top[0]->channels();
  const int top_height = top[0]->height();
  const int top_width = top[0]->width();
  const int bottom_height = bottom[0]->height();
  const int bottom_width = bottom[0]->width();
  const int length = num * top_height * top_width;
  size_t blob_num = this->blobs_.size();
  const Dtype* weightMask = this->blobs_[blob_num - 1]->gpu_data();
  
  caffe_gpu_set(bottom_count, Dtype(0), bottom[0]->mutable_gpu_diff());
  if (this->layer_param_.convolution_param().bias_term()
        && this->param_propagate_down_[1]) {
    const int bias_buffer_count = bias_buffer_.count();
    Dtype* bias_buffer_mutable_data = bias_buffer_.mutable_gpu_data();
    CConvolutionDepthwiseBiasBackward<Dtype>
          // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(bias_buffer_count), CAFFE_CUDA_NUM_THREADS>>>(
        bias_buffer_count, top_diff, num, channels,
        top_height, top_width, bias_buffer_mutable_data);
    const int bias_count = this->blobs_[1]->count();
    const Dtype* bias_buffer_data = bias_buffer_.gpu_data();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    const Dtype* bias_multiplier_data = bias_multiplier_.gpu_data();
    caffe_gpu_gemv(CblasNoTrans, bias_count, length, Dtype(1),
          bias_buffer_data, bias_multiplier_data, Dtype(1), bias_diff);
  }
  if (this->param_propagate_down_[0]) {
    const int weight_buffer_count = weight_buffer_.count();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* weight_buffer_mutable_data = weight_buffer_.mutable_gpu_data();
    CConvolutionDepthwiseWeightBackward<Dtype>
          // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(weight_buffer_count), CAFFE_CUDA_NUM_THREADS>>>(
        weight_buffer_count, top_diff, bottom_data, num, channels,
        top_height, top_width, bottom_height, bottom_width,
        kernel_h_, kernel_w_, stride_h_, stride_w_,
        pad_h_, pad_w_, dilation_h_, dilation_w_, weight_buffer_mutable_data);
    const int weight_count = this->blobs_[0]->count();
    const Dtype* weight_buffer_data = weight_buffer_.gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    const Dtype* weight_multiplier_data = weight_multiplier_.gpu_data();
    caffe_gpu_gemv(CblasNoTrans, weight_count, length, Dtype(1),
          weight_buffer_data, weight_multiplier_data, Dtype(1), weight_diff);
      CCMaskApply<Dtype><<<CAFFE_GET_BLOCKS(this->blobs_[blob_num - 1]->count()),
        CAFFE_CUDA_NUM_THREADS>>>( this->blobs_[blob_num - 1]->count(), weight_diff, weightMask, weight_diff);
      CUDA_POST_KERNEL_CHECK; 			
  
  }
  if (propagate_down[0]) {
    const Dtype* weight_data = this->blobs_[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    CConvolutionDepthwiseBottomBackward<Dtype>
          // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom_count, top_diff, weight_data, num, channels,
        top_height, top_width, bottom_height, bottom_width,
        kernel_h_, kernel_w_, stride_h_, stride_w_,
        pad_h_, pad_w_, dilation_h_, dilation_w_, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CConvolutionDepthwiseLayer);

}  // namespace caffe
